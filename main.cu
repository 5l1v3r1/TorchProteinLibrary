#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <assert.h>

#include "cnpy.h"
#include "cBackboneProteinCUDAKernels.h"

static void DO_CHECK(hipError_t res, const char *fn) {
  if (res != hipSuccess) {
    fprintf(stderr, "Error in %s\n", fn);
    exit(1);
  }
}

#define CHECK(fn) DO_CHECK(fn, #fn)


template<typename T> T *to_gpu(cnpy::NpyArray &a) {
  T *res;
  CHECK(hipMalloc(&res, a.num_bytes()));
  CHECK(hipMemcpy(res, a.data<T>(), a.num_bytes(), hipMemcpyHostToDevice));
  return res;
}

template<typename T> T *load_to_gpu(const char *fname) {
  cnpy::NpyArray tmp = cnpy::npy_load(fname);
  return to_gpu<T>(tmp);
}

static bool close(float f1, float f2) {
  return abs(f1 - f2) <= (1e-08f + (1e-05f * abs(f2)));
}

double diff(timespec start, timespec end)
{
  timespec temp;
  if ((end.tv_nsec-start.tv_nsec)<0) {
    temp.tv_sec = end.tv_sec-start.tv_sec-1;
    temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
  } else {
    temp.tv_sec = end.tv_sec-start.tv_sec;
    temp.tv_nsec = end.tv_nsec-start.tv_nsec;
  }
  return (double)temp.tv_sec + temp.tv_nsec/1000000000.0;
}

#define NLOOPS 100

int main(void) {
  cnpy::NpyArray refGradInput = cnpy::npy_load("gradInputRef.npy");
  float *cpuRefGradInput = (float *)malloc(refGradInput.num_bytes());

  float *gradInput = load_to_gpu<float>("gradInput.npy");
  float *gradOutput = load_to_gpu<float>("gradOutput.npy");
  float *dr_dangle = load_to_gpu<float>("dr_dangle.npy");
  float *input_angles = load_to_gpu<float>("input_angles.npy");
  float *A = load_to_gpu<float>("A.npy");
  int *angles_length = load_to_gpu<int>("angles_length.npy");

  timespec time1, time2;

  cpu_computeDerivativesBackbone(input_angles, dr_dangle, A, angles_length, 64, 650);
  
  cpu_backwardFromCoordsBackbone(gradInput, gradOutput, dr_dangle, angles_length, 64, 650, false);

  CHECK(hipMemcpy(cpuRefGradInput, gradInput, refGradInput.num_bytes(), hipMemcpyDeviceToHost));
  for (int i = 0; i < refGradInput.num_vals; i++) {
    if (!close(cpuRefGradInput[i], refGradInput.data<float>()[i])) {
      printf("[%d]: %12.6f, %12.6f\n", i, cpuRefGradInput[i],
             refGradInput.data<float>()[i]);
    }
  }

  CHECK(hipDeviceSynchronize());
  clock_gettime(CLOCK_MONOTONIC, &time1);

  for (int i = 0; i < NLOOPS; i++) {
    cpu_backwardFromCoordsBackbone(gradInput, gradOutput, dr_dangle, angles_length, 64, 650, false);
  }
  CHECK(hipDeviceSynchronize());
  clock_gettime(CLOCK_MONOTONIC, &time2);

  printf("Took %0.6fs\n", diff(time1, time2)/NLOOPS);
}
