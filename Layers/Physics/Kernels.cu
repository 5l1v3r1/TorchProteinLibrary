#include "hip/hip_runtime.h"
#include <Kernels.h>
#include <stdio.h>

#include <cusp/dia_matrix.h>
#include <cusp/monitor.h>
#include <cusp/precond/diagonal.h>
#include <cusp/krylov/cg.h>

#include <thrust/copy.h>
#include <thrust/transform.h>
#include <thrust/functional.h>


__global__ void partialSumFaces(    float* coords, float* assigned_params, int num_atoms, float* volume, 
                                    int box_size, float res, float stern_size){

    int d = 2;
    int vol_index = threadIdx.x;
	float *single_volume = volume + vol_index * box_size*box_size*box_size;

    float cell_x, cell_y, cell_z;
    float shift_cell_x=0.0, shift_cell_y=0.0, shift_cell_z=0.0;
    float add_sigma = 0.0;
    switch(vol_index){
        case 0:
            shift_cell_x = 0.5;
            break;
        case 1:
            shift_cell_y = 0.5;
            break;
        case 2:
            shift_cell_z = 0.5;
            break;
        case 3:
            add_sigma = stern_size;
            break;
    }
    long cell_idx;

    float x, y, z;
    int x_i, y_i, z_i;
    float r2;


	for(int idx = 0; idx<num_atoms; idx++){
        float sigma = assigned_params[idx] + add_sigma;
		x = coords[3*idx + 0];
		y = coords[3*idx + 1];
		z = coords[3*idx + 2];
		
        x_i = floor(x/res);
		y_i = floor(y/res);
		z_i = floor(z/res);
		for(int i=x_i-d; i<=(x_i+d);i++){
			for(int j=y_i-d; j<=(y_i+d);j++){
				for(int k=z_i-d; k<=(z_i+d);k++){
					if( (i>=0 && i<box_size) && (j>=0 && j<box_size) && (k>=0 && k<box_size) ){
						cell_idx = k + j*box_size + i*box_size*box_size;
						cell_x = i*res + shift_cell_x;
                        cell_y = j*res + shift_cell_y;
                        cell_z = k*res + shift_cell_z;
                        r2 = (x-cell_x)*(x-cell_x) + (y-cell_y)*(y-cell_y) + (z-cell_z)*(z-cell_z);
						single_volume[cell_idx] += exp(-r2/(sigma*sigma));
					}
				}
			}
		}
	}
}

__global__ void sumCells(   float* coords, float* assigned_params, int num_atoms, float* volume, 
                            int box_size, float res){
	for(int idx = 0; idx<num_atoms; idx++){
		float x = coords[3*idx + 0],
            y = coords[3*idx + 1],
            z = coords[3*idx + 2];
		int x_i = floor(x/res);
		int y_i = floor(y/res);
		int z_i = floor(z/res);
        int cell_idx = x_i + y_i*box_size + z_i*box_size*box_size;
		volume[cell_idx] += assigned_params[idx];
	}
}

void gpu_computePartialSumFaces(	float *coords,
                                    float *assigned_params,
                                    int num_atoms, 
                                    float *volume,
                                    int box_size,
                                    float res,
                                    float stern_size){
	partialSumFaces<<<1, 4>>>(coords, assigned_params, num_atoms, volume, box_size, res, stern_size);
}

void gpu_computeSumCells(	float *coords,
                            float *assigned_params,
                            int num_atoms, 
                            float *volume,
                            int box_size,
                            float res){
	sumCells<<<1, 1>>>(coords, assigned_params, num_atoms, volume, box_size, res);
}

struct saxpy_functor
{
    const float mul;
    saxpy_functor(float _mul):mul(_mul){}

    __host__ __device__
    float operator()(const float& x, const float& y) const
    { 
        return mul * x + y;
    }
};


void gpu_computePhi( float *Q, float *Eps, float *Phi, int box_size, float res, float kappa02){
    size_t surf_size = box_size*box_size; 
    size_t vol_size = box_size*box_size*box_size;
    size_t num_nonzero = 7*vol_size - 2*surf_size - 2*box_size - 2;

    cusp::dia_matrix<size_t, float, cusp::device_memory> A(vol_size, vol_size, num_nonzero, 7);
    cusp::array1d<float, cusp::device_memory> phi(vol_size, 0.0);
    cusp::array1d<float, cusp::device_memory> q(vol_size, 0.0);
    A.diagonal_offsets[0] = -box_size*box_size;
    A.diagonal_offsets[1] = -box_size;
    A.diagonal_offsets[2] = -1;
    A.diagonal_offsets[3] = 0;
    A.diagonal_offsets[4] = 1;
    A.diagonal_offsets[5] = box_size;
    A.diagonal_offsets[6] = box_size*box_size;
    
    thrust::fill(A.values.values.begin(), A.values.values.end(), 0.0);
    
    thrust::device_ptr<float> ei_begin(Eps);
    thrust::device_ptr<float> ej_begin(Eps + vol_size);
    thrust::device_ptr<float> ek_begin(Eps + 2*vol_size);
    thrust::device_ptr<float> lambda_begin(Eps + 3*vol_size);
    thrust::device_ptr<float> q_begin(Q);
    thrust::device_ptr<float> Phi_begin(Phi);
    
    //Lower diagonals
    thrust::transform(  ei_begin, ei_begin + vol_size - surf_size, A.values.column(0).begin() + surf_size, 
                        A.values.column(0).begin() + surf_size, saxpy_functor(-1.0/res*res));
    thrust::transform(ej_begin, ej_begin + vol_size - box_size, A.values.column(1).begin() + box_size, 
                        A.values.column(1).begin() + box_size, saxpy_functor(-1.0/res*res));
    thrust::transform(ek_begin, ek_begin + vol_size - 1, A.values.column(2).begin() + 1, 
                        A.values.column(2).begin() + 1, saxpy_functor(-1.0/res*res));

    //Upper diagonals
    thrust::transform(ek_begin, ek_begin + vol_size - 1, A.values.column(4).begin(), 
                        A.values.column(4).begin(), saxpy_functor(-1.0/res*res));
    thrust::transform(ej_begin, ej_begin + vol_size - box_size, A.values.column(5).begin(), 
                        A.values.column(5).begin(), saxpy_functor(-1.0/res*res));
    thrust::transform(ei_begin, ei_begin + vol_size - surf_size, A.values.column(6).begin(), 
                        A.values.column(6).begin(), saxpy_functor(-1.0/res*res));
    
    //Diagonal
    thrust::transform(ei_begin, ei_begin + vol_size, A.values.column(3).begin(), 
                        A.values.column(3).begin(), saxpy_functor(1.0/res*res));
    thrust::transform(ej_begin, ej_begin + vol_size, A.values.column(3).begin(), 
                        A.values.column(3).begin(), saxpy_functor(1.0/res*res));
    thrust::transform(ek_begin, ek_begin + vol_size, A.values.column(3).begin(), 
                        A.values.column(3).begin(), saxpy_functor(1.0/res*res));
    
    //diagonal shifted
    thrust::transform(ei_begin, ei_begin + vol_size - surf_size, A.values.column(3).begin() + surf_size, 
                        A.values.column(3).begin() + surf_size, saxpy_functor(1.0/res*res));
    thrust::transform(ej_begin, ej_begin + vol_size - box_size, A.values.column(3).begin() + box_size,
                        A.values.column(3).begin() + box_size, saxpy_functor(1.0/res*res));
    thrust::transform(ek_begin, ek_begin + vol_size - 1, A.values.column(3).begin() + 1,
                        A.values.column(3).begin() + 1, saxpy_functor(1.0/res*res));
    
    //ionic term
    thrust::transform(lambda_begin, lambda_begin + vol_size, A.values.column(3).begin(), 
                        A.values.column(3).begin(), saxpy_functor(kappa02));

    //charge
    thrust::transform(q_begin, q_begin + vol_size, q.begin(), q.begin(), saxpy_functor(1.0/res*res*res));

    cusp::monitor<float> monitor(q, 1000, 1e-3, 0.0, true);
    monitor.set_verbose();
    cusp::precond::diagonal<float, cusp::device_memory> M(A);
    cusp::krylov::cg(A, phi, q, monitor);
    monitor.print();

    thrust::copy(phi.begin(), phi.end(), Phi);

       
}