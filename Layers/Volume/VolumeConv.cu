#include "hip/hip_runtime.h"
#include "VolumeConv.h"
#include <hipfft/hipfft.h>

#define WARP_SIZE 32

__global__ void conjMul(hipfftComplex *c_volume1, hipfftComplex *c_volume2, hipfftComplex *c_output, int batch_size, int volume_size, bool conj){
	uint batch_idx = blockIdx.x;
    uint warp_idx = blockIdx.y;
    uint thread_idx = threadIdx.x;
    int reduced_volume_size = (volume_size/2 + 1);
    uint vol = volume_size*volume_size*reduced_volume_size;
    uint full_vol = volume_size*volume_size*volume_size;
    uint memory_idx = batch_idx*vol + warp_idx*WARP_SIZE + thread_idx;

    if( (warp_idx*WARP_SIZE + thread_idx) >= vol) //out of volume
        return;
    REAL re, im;
    if(conj){
        re = c_volume1[memory_idx].x * c_volume2[memory_idx].x + c_volume1[memory_idx].y * c_volume2[memory_idx].y;
        im = -c_volume1[memory_idx].x * c_volume2[memory_idx].y + c_volume1[memory_idx].y * c_volume2[memory_idx].x;
    }else{
        re = c_volume1[memory_idx].x * c_volume2[memory_idx].x - c_volume1[memory_idx].y * c_volume2[memory_idx].y;
        im = c_volume1[memory_idx].x * c_volume2[memory_idx].y + c_volume1[memory_idx].y * c_volume2[memory_idx].x;
    }
    // printf("%f %f \n", c_volume1[memory_idx].x, c_volume2[memory_idx].y);
    c_output[memory_idx].x = re*2.0/full_vol;
    c_output[memory_idx].y = im*2.0/full_vol;

}

__global__ void mulInPlace(REAL *d_volume1, REAL *d_volume2, int batch_size, int volume_size){
	uint batch_idx = blockIdx.x;
    uint warp_idx = blockIdx.y;
    uint thread_idx = threadIdx.x;
    uint vol = volume_size*volume_size*volume_size;
    uint memory_idx = batch_idx*vol + warp_idx*WARP_SIZE + thread_idx;

    if( (warp_idx*WARP_SIZE + thread_idx) >= vol) //out of volume
        return;

    d_volume1[memory_idx] *= d_volume2[memory_idx];
}

void cpu_VolumeConv(REAL *d_volume1,  REAL *d_volume2,  REAL *d_output, int batch_size, int volume_size){
    hipfftHandle plan_fwd, plan_bwd;
    hipfftComplex *c_volume1, *c_volume2, *c_output;
    int reduced_volume_size = (volume_size/2 + 1);
    hipMalloc((void**)&c_volume1, sizeof(hipfftComplex)*batch_size*volume_size*volume_size*reduced_volume_size);
    hipMalloc((void**)&c_volume2, sizeof(hipfftComplex)*batch_size*volume_size*volume_size*reduced_volume_size);
    hipMalloc((void**)&c_output, sizeof(hipfftComplex)*batch_size*volume_size*volume_size*volume_size);

    int dimensions_real[] = {volume_size, volume_size, volume_size};
    int dimensions_complex[] = {volume_size, volume_size, reduced_volume_size};
    int batch_volume_real = volume_size*volume_size*volume_size;
    int batch_volume_complex = volume_size*volume_size*reduced_volume_size;
    int inembed[] = {volume_size, volume_size, volume_size};
    int onembed[] = {volume_size, volume_size, reduced_volume_size};
    hipfftPlanMany(&plan_fwd, 3, dimensions_real, 
                    inembed, 1, batch_volume_real, 
                    onembed, 1, batch_volume_complex,
                    HIPFFT_R2C, batch_size);

    hipfftPlanMany(  &plan_bwd, 3, dimensions_complex, 
                    onembed, 1, batch_volume_complex, 
                    inembed, 1, batch_volume_real,
                    HIPFFT_C2R, batch_size);
    hipfftExecR2C(plan_fwd, d_volume1, c_volume1);
    hipfftExecR2C(plan_fwd, d_volume2, c_volume2);

    dim3 dim_special(batch_size, batch_volume_complex/WARP_SIZE + 1);
	conjMul<<<dim_special, WARP_SIZE>>>(c_volume1, c_volume2, c_output, batch_size, volume_size, true);

    hipfftExecC2R(plan_bwd, c_output, d_output);
    
    hipfftDestroy(plan_fwd);
    hipfftDestroy(plan_bwd);
    hipFree(c_volume1);
    hipFree(c_volume2);
    hipFree(c_output);

}


